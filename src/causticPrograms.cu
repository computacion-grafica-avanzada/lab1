#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <hip/hip_runtime.h>

#include "LaunchParams.h"
#include "gdt/random/random.h"

#include <hip/hip_runtime.h>
#include "Utils.h"
#include "ONB.h"

using namespace osc;

namespace osc {

	typedef gdt::LCG<16> Random;

	/*! launch parameters in constant memory, filled in by optix upon
		optixLaunch (this gets filled in from the buffer we pass to
		optixLaunch) */
	extern "C" __constant__ LaunchParams optixLaunchParams;

	/*! per-ray data now captures random number generator, so programs
		can access RNG state */

	struct PhotonPRD {
		Random random;
		vec3f power;
		unsigned int depth;
	};

	extern "C" __global__ void __closesthit__caustic()
	{
	}

	extern "C" __global__ void __anyhit__caustic()
	{
	}

	extern "C" __global__ void __miss__caustic()
	{
	}

	extern "C" __global__ void __raygen__renderCaustic()
	{
	}

}
